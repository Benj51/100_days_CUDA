#include <iostream>
#include "hip/hip_runtime.h"
#include <stdlib.h>

__global__ 
void KernelAdd(float *A,float *B,float *C,int n){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i<n){
        C[i]= A[i]+B[i];
    }
}

void VecADD(float *A_h,float *B_h,float *C_h,int n){
    int size = n*sizeof(float);
    float *A_d, *B_d, *C_d;

    hipMalloc((void**)&A_d,size);
    hipMalloc((void**)&B_d,size);
    hipMalloc((void**)&C_d,size);

    hipMemcpy(A_d,A_h,size,hipMemcpyHostToDevice);
    hipMemcpy(B_d,B_h,size,hipMemcpyHostToDevice);

    KernelAdd <<<ceil(n/256.0),256>>>(A_d, B_d, C_d,n);

    hipMemcpy(C_h,C_d,size,hipMemcpyDeviceToHost);

    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);

}

int main() {

int n = 5;

float A[] = {1.0f, 3.0f, 5.0f, 7.0f, 9.0f};
float B[] = {2.0f, 4.0f, 6.0f, 8.0f, 10.0f};
float C[5];

VecADD(A,B,C,n);

 printf("Resultats:\n");
int i;
for(i = 0; i < n; i++) {
    printf("%f + %f = %f\n", A[i], B[i], C[i]);
    }

     return 0;
}