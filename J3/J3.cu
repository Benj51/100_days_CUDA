#include <iostream>
#include "hip/hip_runtime.h"
#include <stdlib.h>

_global__ 
void KernelMat_Add(float *M,float*N,float *P,int dim){

    int row = blockDim.y*blockIdx.y +threadIdx.y;
    int column = blockDim.x*blockIdx.x +threadIdx.x;


    if((row<dim)&&(column<dim)){
        P[row*dim + column] = M[row*dim + column] + N[row*dim + column];
    }


}

void MatADD(float *M_h,float *N_h,float *P_h,int n){
    int size = n*n*sizeof(float);
    float *M_d, *N_d, *P_d;

    hipMalloc((void**)&M_d,size);
    hipMalloc((void**)&N_d,size);
    hipMalloc((void**)&P_d,size);

    hipMemcpy(M_d,M_h,size,hipMemcpyHostToDevice);
    hipMemcpy(N_d,N_h,size,hipMemcpyHostToDevice);

    dim3 dimGrid((n + 15) / 16, (n + 15) / 16, 1);
    dim3 dimBlock(16,16,1);

    KernelMat_Add <<<dimGrid,dimBlock>>>(M_d, N_d, P_d,n);

    hipMemcpy(P_h,P_d,size,hipMemcpyDeviceToHost);

    hipFree(M_d);
    hipFree(N_d);
    hipFree(P_d);

}