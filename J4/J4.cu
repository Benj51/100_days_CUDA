#include <iostream>
#include "hip/hip_runtime.h"
#include <stdlib.h>


__global__ 
void Kernel_MatMul(float *M_h,float *N_h,float *P_h,int n){

    int row = blockDim.y * blockIdx.y +threadIdx.y;
    int column = blockDim.x * blockIdx.x +threadIdx.x;

    if((row<n)&&(column<n)){
        float dot_product = 0.0f;
        for(int k = 0; k<n; k++){
            dot_product += M_h[row*k+column] * N_h[row*k+column];      
        }
        P_h[row*n+column]= dot_product;
    }

}

void MatMul(float *M_h,float *N_h,float *P_h,int n){
    int size = n*n*sizeof(float);
    float *M_d, *N_d, *P_d;

    hipMalloc((void**)&M_d,size);
    hipMalloc((void**)&N_d,size);
    hipMalloc((void**)&P_d,size);

    hipMemcpy(M_d,M_h,size,hipMemcpyHostToDevice);
    hipMemcpy(N_d,N_h,size,hipMemcpyHostToDevice);

    dim3 dimGrid((n + 15) / 16, (n + 15) / 16, 1);
    dim3 dimBlock(16,16,1);

   Kernel_MatMul <<<dimGrid,dimBlock>>>(M_d, N_d, P_d,n);

    hipMemcpy(P_h,P_d,size,hipMemcpyDeviceToHost);

    hipFree(M_d);
    hipFree(N_d);
    hipFree(P_d);

}
